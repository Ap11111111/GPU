#include<hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
// #include<bits/stdc++.h>
#include <stdlib.h>
#include <string.h>
using namespace std;
__global__ void mul( int *ro, int *co ,int *aa,int *fv,int v,int x,int *res,int ex){
    int row=blockIdx.x*blockDim.x+threadIdx.x;
    if(row<ex){
        int rinitial=ro[row];
        int rend=ro[row+1];
        int r_new=aa[row];
         printf("%d\t %d\n",r_new,row);
        double  sum,offset;
       // sum=0;
        for(int j=0;j<x;j++){
          sum=0;
          for(int h=rinitial;h<rend;h++){  
            offset=fv[co[h]*x+j];
           printf("offset %lf\n",offset); 
            //printf("va[h] %d\n",va[h]);
               
            sum+=offset;
            //printf("sum %lf\t %d\n",sum,row);
          }
         // printf("%lf\t",sum);
          res[r_new*x+j]=sum;
        // printf("res %lf\t tid %d\n",res[r_new*x+j],row);
        }
    }
    //printf("res %lf\t tid %d\n",res[0],res[1]);
}

int main()
{ 
  
  int vertex=10;
  int v=vertex;
  char buffer[8090];
  char *record,*line;
  int i=0,j=0;
  // int begn=1;
  // int last=v+1;
  int ex=0;
  int rx=0;
  int incrr=ex+1;
  int t=0;
  int n=0,m=1,f=2,nnz=INT_MAX;
  int z=1;
  int d=0;
  int q=0;
  int s=0;
  //int c=0;
  //int* mat = new int[v * v];
   int* mat = (int*)malloc(v*v*sizeof(int));
  int* rowptr = (int*)malloc((v+1)*sizeof(int));
  int* rowptr_nxt=(int*)malloc(incrr*sizeof(int));
  int* ar_nxt=(int*)malloc(ex*sizeof(int));
  int* mx = (int*)malloc(v*v*sizeof(int));
  double *rowptr_new=(double*)malloc((v+1)*sizeof(double));
  int* b=(int*)malloc(v*f*sizeof(int));
  int* g=(int*)malloc(v*f*sizeof(int));
  int* colindex=(int*)malloc(nnz*sizeof(int));
  int* colindex_new=(int*)malloc(nnz*sizeof(int));
  int* access=(int*)malloc(v*sizeof(int));
  int* acc=(int*)malloc(v*sizeof(int));
  int* n_o_p=(int*)malloc(v*sizeof(int));
  int* par=(int*)malloc(v*sizeof(int));
  double *rpt;
  int *cdx,*b_f;
  int *res;
  int *rrpt,*aa_ne;
  hipMalloc(&res, v*f*sizeof(int));
  hipMalloc(&rpt, (v+1)*sizeof(double));
  hipMalloc(&b_f, v*f* sizeof(int));
   //cudaMalloc(&res,v*f*sizeof(double));
   //cudaMalloc(&vls, nnz * sizeof(int));
FILE *fstream = fopen("ex_3.csv","r");
  if(fstream == NULL)
    {
      printf("\n file opening failed ");
      return -1 ;
    }
  
   
  while((line=fgets(buffer,sizeof(buffer),fstream))!=NULL)
  {
    record = strtok(line,";");
    while(record != NULL)
    {

      int x =0;
      for(int k=0;k<strlen(record);k+=2)
      {
         
        mat[i * v + x] = record[k]-'0';
        x++;
      }
      
      record = strtok(NULL,",");
      
    }
     
    i++ ;
  
    // if (i == v) break;
  }
  for(int i=0;i<v;i++)
  {
    for(int j=0;j<v;j++)
    {
     mx[i * v + j] = mat[i * v + j];
    }
  }
   
  const char* filename="partitions.csv";
  FILE *file = fopen(filename,"r");
  if(file==NULL)
    {
      printf("not open");
      exit(0);
    }    

  int ind;
  int p=0;
  while(fscanf(file,"%d",&ind)!=EOF)
    {
      access[p]=ind;
      p++;
      // if(p == v)
      //   break;
    } 
  for(int i=0;i<p;i++)
  {
    acc[q]= access[i];
    q++;
    // if(q==v)
    //   break;
  }
  //////////////////////////////////////////////////////////////////////////////////////
  const char* filenam="ex_3_3.csv";
  FILE *fil = fopen(filenam,"r");
  if(fil==NULL)
    {
      printf("not open");
      exit(0);
    }    

  int in;
  int a=0;
  while(fscanf(fil,"%d",&in)!=EOF)
    {
      n_o_p[a]=in;
      a++;
      // if(p == v)
      //   break;
    } 
  for(int i=0;i<a;i++)
  {
    par[s]= n_o_p[i];
    s++;
    // if(q==v)
    //   break;
  }
   ///////////////////////////////////////////////////////////////////////////////////////// 

   // printf("calculate the column indices\n");
  for(int i=0;i<v;i++)
  {
    for(int j=0;j<v;j++)
    {
      if(mx[i * v + j]==1)
      {
          colindex[n++]=j+1;
      }
    }
  }
  /////////////////////////////////////////////////
  // for(int i=0;i<n;i++){
  //   printf("%d",colindex[i]);
  // }
 // printf("%d",n);
//printf("feature matrix\n");
  for(int i=0;i<v*f;i++)
  {
     b[i] = 1;
    //  c_v++;
    //  printf("%d\n",c_v);
  }
  
  
//printf("new rowptr is\n");
  for( int i=0;i<q;i++)
  {
    // printf("%d\n",i);
    //  printf("i: %d\n",i);
    for( int j=0;j<v;j++)
    {
      if(j==acc[i]-1)
      {
        // printf("j: %d\n",j);
        int  rowval_new=0;
        for( int k=0;k<v;k++)
        {
          // printf("k: %d\n",j);
          if(mx[j * v + k] == 1)
          {
            rowval_new++;
            // printf("j: %d\n",j);
          } 
            
          rowptr_new[0]=0;
          rowptr_new[z]=rowval_new+rowptr_new[z-1];
        }
          
        z++;
        // if(z==v)
        // break;
      }
    }
  }
 // printf("%d",z);
  //printf("the rowptr array corresponding \n");
  // for(int i=0;i<z;i++){
  //    printf("%lf\n",rowptr_new[i]);
  //    }
  //printf("new colindex \n ");
  for(int i=0;i<q;i++)
  {
    // printf("%d\n",i);
    for(j=0;j<v;j++)
    {
      if(j==acc[i]-1)
      {
        for(int k=0;k<v;k++)
        {
          if(mx[j * v + k]==1)
          {
            colindex_new[d++]=k+1;
                    
          }
                
        }
            
      }
      
    }
  }
 ///////////////////
    // for( int i=0;i<d;i++){
    //     printf("%d\n",colindex_new[i]);
    //  }
  hipMalloc(&cdx, d * sizeof(int));
  
  hipMemcpy(cdx, colindex_new, d * sizeof(int),
                                  hipMemcpyHostToDevice ) ;
  hipMemcpy( b_f, b, v*f* sizeof(int),
                                  hipMemcpyHostToDevice );
  //printf("calculate the rowpointer\n");
  for(int i=0;i<v;i++)
  {
    int rowval=0;
    for(int j=0;j<v;j++)
    {
      if(mx[i * v + j]==1)
      {
        rowval++;
      }
      rowptr[0]=0;
      rowptr[m]=rowval+rowptr[m-1];
        
    }
    m++;
  }
 int c=0;
  while (c<s)
  {
    int  noz=INT_MAX;
    ex=par[c];
      //printf("%d",c);
    t=rx;
    rx+=ex;
    int sum=0;
    for(int i=t+1; i<=rx; i++)
    {
      if(i==rx)
      sum= rowptr_new[rx]-rowptr_new[t];
    }
     //printf("%d",sum);
    noz=noz-sum;
    if(noz>sum)
    {

      //    printf("send value till %d",rx);.
      //    printf("next row_ptr value is");
      rowptr_nxt[0]=rowptr_new[t];
      int r=1;
      for(int i=t+1;i<=rx;i++)
      {
        rowptr_nxt[r] = rowptr_new[i];
        r++;
      }
      //////////////////////////////////////
      // for(int i=0;i<r;i++)
      // {
      // printf("%d",rowptr_nxt[i]);
      // }
        int y=0;
        for(int i=t;i<rx;i++)
        {
          ar_nxt[y]= acc[i];
          y++;
        }
        //////////////////////////////////////////////////////////////////////////////////////
        // for(int i=0;i<y;i++)
        // printf("%d",ar_nxt[i]);
        
        hipMalloc(&rrpt,  incrr* sizeof(int));
        
        hipMalloc(&aa_ne, ex* sizeof(int));
        
    
        hipMemcpy( rrpt, rowptr_nxt, incrr * sizeof(int), hipMemcpyHostToDevice );
              
        hipMemcpy(aa_ne , ar_nxt, ex * sizeof(int),hipMemcpyHostToDevice );
                        
               
        //  float elapsed_time;
        // cudaEvent_t start, stop;
        // cudaEventCreate(&start);
        // cudaEventCreate(&stop);

        // cudaEventRecord(start);
        //  mul<<<v/ex,ex>>>(rrpt,cdx,aa_ne,b_f,v,f,res,ex);
          mul<<<1,ex>>>(rrpt,cdx,aa_ne,b_f,v,f,res,ex);
          
        //    cudaEventRecord(stop);
        //    cudaDeviceSynchronize();
        //   cudaEventSynchronize(stop);
        //   cudaEventElapsedTime(&elapsed_time, start, stop);

        //  printf("\nRunning time:  %.4f ms\n", elapsed_time);
        hipDeviceSynchronize();
          
         //begn+=ex;
        
      //   c++;
      //  printf("%d",c);
      }
    else
    break;
    c++;
  }

  hipMemcpy(g,res,v*f*sizeof(int),hipMemcpyDeviceToHost);
   //for(i=0; i<v; i++)
     //printf("i: g[0] %lf\t, g[1] %lf\n",g[i*f],g[i*f+1]);
     ////////////////////////////
  //  for(int i=0; i<v; i++)
  //      for(int j=0;j<f;j++)
  //        printf("%d\n", g[i*f+j]);
         /////////////////////////
    // cudaFree(res);
  //  fclose(fopen("result.txt", "w"));

  //   char buf[100];
  //   double *temp1 = (double*) malloc(sizeof(double));
  //   double *temp2 = (double*) malloc(sizeof(double));
    
  //   FILE *results = fopen("result.txt", "a");
  
      // for(int i=0;i<v*f;i+=2){}
      //    printf("result %lf \t",g[i*f+j]);
  //         cudaMemcpy(temp1,res+(i),sizeof(double),cudaMemcpyDeviceToHost);
  //         cudaMemcpy(temp2,res+(i+1),sizeof(double),cudaMemcpyDeviceToHost);
  //        snprintf(buf, 50, "Result : %d -> %lf\t%lf", (i/2), *temp1, *temp2);
  //        fprintf(results, "%s\n", buf);
  //     }
  //     printf("end\n");
  //     fclose(results);
}
